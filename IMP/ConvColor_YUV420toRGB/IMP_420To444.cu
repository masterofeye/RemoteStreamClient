#include "hip/hip_runtime.h"
#ifndef IMP_420To444_CU
#define IMP_420To444_CU

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>


__global__ void kernel420To444(uint8_t *pYUV420Array, uint8_t *pArrayFull, int iWidth, int iHeight, int iPitch)
{
    int iPosY = blockIdx.y * blockDim.y + threadIdx.y;
    int iPosX = blockIdx.x * blockDim.x + threadIdx.x;
    int iPos420 = iPosY * iPitch + iPosX;
    int iOffset = iHeight * iPitch;
    int iDepth = 3;
    //              |        y Pos        | + |   x Pos    | + |z|
    int iPos444 = iPosY * iDepth * iWidth + iDepth * iPosX +  0; 

    if (iPosX < iWidth)
    {
        pArrayFull[iPos444] = pYUV420[iPos420];
        //int val = pYUV420Array[0];

        if (iPosY < iHeight / 2)
        {
            if (iPosX % 2 == 0){
                pArrayFull[(2 * iPosY) * iDepth * iWidth + iDepth * iPosX + 1] = pYUV420[iOffset + iPos420];
                pArrayFull[(2 * iPosY) * iDepth * iWidth + iDepth * (iPosX + 1) + 1] = pYUV420[iOffset + iPos420];
                pArrayFull[((2 * iPosY) + 1) * iDepth * iWidth + iDepth * iPosX + 1] = pYUV420[iOffset + iPos420];
                pArrayFull[((2 * iPosY) + 1) * iDepth * iWidth + iDepth * (iPosX + 1) + 1] = pYUV420[iOffset + iPos420];
            }
            else 
            {
                pArrayFull[(2 * iPosY) * iDepth * iWidth + iDepth * iPosX + 2] = pYUV420[iOffset + iPos420];
                pArrayFull[(2 * iPosY) * iDepth * iWidth + iDepth * (iPosX + 1) + 2] = pYUV420[iOffset + iPos420];
                pArrayFull[((2 * iPosY) + 1) * iDepth * iWidth + iDepth * iPosX + 2] = pYUV420[iOffset + iPos420];
                pArrayFull[((2 * iPosY) + 1) * iDepth * iWidth + iDepth * (iPosX + 1) + 2] = pYUV420[iOffset + iPos420];
            }
        }
    }
}

extern "C" void IMP_420To444(uint8_t *pYUV420Array, uint8_t *pArrayFull, int iWidth, int iHeight, size_t pitchY)
{
    dim3 block(32, 16, 1);
    dim3 grid(pitchY / block.x, iHeight / block.y, 1);

    //interleaved to plane
    kernel420To444 << <grid, block >> >(pYUV420Array, pArrayFull, iWidth, iHeight, (int)pitchY);
}

#endif